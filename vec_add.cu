
#include <hip/hip_runtime.h>
#include <iostream>

#define PROJECT_NAME "learn-cuda"
#define N 64

__global__ void VecAdd(float *A, float *B, float *C) {
  int i = threadIdx.x;
  C[i] = A[i] + B[i];
}

int main() {
  float *A = new float[N];
  float *B = new float[N];
  float *C = new float[N];
  for (int i = 0; i < N; i++) {
    A[i] = i;
    B[i] = i;
  }

  float *d_A, *d_B, *d_C;

  // If we do not malloc, the results will all be zeros.
  hipMalloc((void **)&d_A, N * sizeof(float));
  hipMalloc((void **)&d_B, N * sizeof(float));
  hipMalloc((void **)&d_C, N * sizeof(float));

  hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);

  VecAdd<<<1, N>>>(d_A, d_B, d_C);
  hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

  for (int i = 0; i < N; i++) {
    std::cout << C[i] << std::endl;
  }

  delete[] A;
  delete[] B;
  delete[] C;
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  return 0;
}
